#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x, float* y)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  printf("tid - %d\n", tid);

  // x[tid] = (float) threadIdx.x;
  x[tid] += (float) y[tid];

}


//
// main code
//

int main(int argc, const char **argv)
{
  float *h_x, *d_x, *h_y, *d_y;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  for (int i=0; i<nsize; i++) 
    h_x[i] = i;

  h_y = (float *)malloc(nsize*sizeof(float));
  for (int i=0; i<nsize; i++) 
    h_y[i] = i;

  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_y, nsize*sizeof(float)));

  checkCudaErrors( hipMemcpy(d_x,h_x,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_y,h_y,nsize*sizeof(float),
                 hipMemcpyHostToDevice) );


  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_x, d_y);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );
  // checkCudaErrors( hipMemcpy(h_y,d_y,nsize*sizeof(float),
                //  hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  checkCudaErrors(hipFree(d_x));
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
