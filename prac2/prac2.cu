#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2, a, b, c;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////

__global__ void expression_calc(float *d_z, float *d_avg) {
  float sum = 0.0f;
  int ind = threadIdx.x + N*blockIdx.x*blockDim.x, z;

  for (int n=0; n<N; n++) {
    z = d_z[ind];
    sum += a*z*z + b+z + c;
    ind += blockDim.x;
  }

  d_avg[threadIdx.x + blockIdx.x*blockDim.x] = sum/N;
}


__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x; // 2*N because each iteration uses two random numbers

  // version 2
  // ind = 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    y2   = rho*y1 + alpha*d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array

  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int     NPATH=960000, h_N=200;
  int K = 2*h_N*NPATH;

  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2, h_a, h_b, h_c;
  float  *h_v, *h_avg, *d_v, *d_z, *d_avg;
  double  sum1, sum2; // sum is done in double precision for accuracy

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);
  h_avg = (float *)malloc(sizeof(float)*(K/200));

  // checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_avg, sizeof(float)*(K/200)) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  h_a = 1.0f;
  h_a = 2.0f;
  h_a = 3.0f;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(a),    &h_a,    sizeof(h_a)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(b),    &h_b,    sizeof(h_b)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(c),    &h_c,    sizeof(h_c)) );

  // random number generation

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );

  hipEventRecord(start);
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f) );
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, 2.0*h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);
  // pathcalc<<<NPATH/128, 128>>>(d_z, d_v);
  expression_calc<<<K/(200*128), 128>>>(d_z, d_avg);

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("pathcalc execution failed\n");
  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  // checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
  //                  hipMemcpyDeviceToHost) );
  
  checkCudaErrors( hipMemcpy(h_avg, d_avg, sizeof(float)*(K/200),
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) { //sum is done on host. this can be done on device
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  for (int i=0; i<(K/200); i++) {
    sum1 += h_avg[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f\n\n",
	 sum1/(K/200));

  // printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	//  sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  // checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );
  checkCudaErrors( hipFree(d_avg) );


  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
